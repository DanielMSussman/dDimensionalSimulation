#include "hip/hip_runtime.h"
#include "utilities.cuh"

/*! \file utilities.cu
  defines kernel callers and kernels for some simple GPU array calculations

 \addtogroup utilityKernels
 @{
 */

/*!
add the first N elements of array and put it in output[helperIdx]
*/
__global__ void gpu_serial_reduction_kernel(scalar *array, scalar *output, int helperIdx,int N)
    {
    scalar ans = 0.0;
    for (int i = 0; i < N; ++i)
        ans += array[i];
    output[helperIdx] = ans;
    return;
    };

/*!
perform a block reduction, storing the partial sums of input into output
*/
__global__ void gpu_parallel_block_reduction_kernel(scalar *input, scalar *output,int N)
    {
    extern __shared__ scalar sharedArray[];

    unsigned int tidx = threadIdx.x;
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    //load into shared memory and synchronize
    if(i < N)
        sharedArray[tidx] = input[i];
    else
        sharedArray[tidx] = 0.0;
    __syncthreads();

    //reduce
    for (int s = blockDim.x/2; s>0; s>>=1)
        {
        if (tidx < s)
            sharedArray[tidx] += sharedArray[tidx+s];
        __syncthreads();
        };
    //write to the correct block of the output array
    if (tidx==0)
        output[blockIdx.x] = sharedArray[0];
    };

/*!
a slight optimization of the previous block reduction, c.f. M. Harris presentation
*/
__global__ void gpu_parallel_block_reduction2_kernel(scalar *input, scalar *output,int N)
    {
    extern __shared__ scalar sharedArray[];

    unsigned int tidx = threadIdx.x;
    unsigned int i = 2*blockDim.x * blockIdx.x + threadIdx.x;

    scalar sum;
    //load into shared memory and synchronize
    if(i < N)
        sum = input[i];
    else
        sum = 0.0;
    if(i + blockDim.x < N)
        sum += input[i+blockDim.x];

    sharedArray[tidx] = sum;
    __syncthreads();

    //reduce
    for (int s = blockDim.x/2; s>0; s>>=1)
        {
        if (tidx < s)
            sharedArray[tidx] = sum = sum+sharedArray[tidx+s];
        __syncthreads();
        };
    //write to the correct block of the output array
    if (tidx==0)
        output[blockIdx.x] = sum;
    };

/*!
  A function of convenience...zero out an array on the device
  */
__global__ void gpu_zero_array_kernel(dVec *arr,
                                              int N)
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    for (int dd = 0; dd < DIMENSION; ++dd)
        arr[idx].x[dd] = 0.0;
    return;
    };
/*!
  A function of convenience...zero out an array on the device
  */
__global__ void gpu_zero_array_kernel(scalar *arr,
                                              int N)
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;

    arr[idx] = 0.;
    return;
    };
/*!
  A function of convenience...zero out an array on the device
  */
__global__ void gpu_zero_array_kernel(unsigned int *arr,
                                              int N)
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;

    arr[idx] = 0;
    return;
    };

/*!
  A function of convenience...zero out an array on the device
  */
__global__ void gpu_zero_array_kernel(int *arr,
                                      int N)
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;

    arr[idx] = 0;
    return;
    };

/////
//Kernel callers
///

bool gpu_zero_array(dVec *arr,
                    int N
                    )
    {
    //optimize block size later
    unsigned int block_size = 128;
    if (N < 128) block_size = 16;
    unsigned int nblocks  = N/block_size + 1;

    gpu_zero_array_kernel<<<nblocks, block_size>>>(arr,
                                                    N
                                                    );
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }

bool gpu_zero_array(unsigned int *arr,
                    int N
                    )
    {
    //optimize block size later
    unsigned int block_size = 128;
    if (N < 128) block_size = 16;
    unsigned int nblocks  = N/block_size + 1;

    gpu_zero_array_kernel<<<nblocks, block_size>>>(arr,
                                                    N
                                                    );
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }

bool gpu_zero_array(scalar *arr,
                    int N
                    )
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 16;
    unsigned int nblocks  = N/block_size + 1;

    gpu_zero_array_kernel<<<nblocks, block_size>>>(arr,
                                                    N
                                                    );
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }
bool gpu_zero_array(int *arr,
                    int N
                    )
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 16;
    unsigned int nblocks  = N/block_size + 1;

    gpu_zero_array_kernel<<<nblocks, block_size>>>(arr,
                                                    N
                                                    );
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }

/*!
a two-step parallel reduction algorithm that first does a partial sum reduction of input into the
intermediate array, then launches a second kernel to sum reduce intermediate into output[helperIdx]
\param input the input array to sum
\param intermediate an array that input is block-reduced to
\param output the intermediate array will be sum reduced and stored in one of the components of output
\param helperIdx the location in output to store the answer
\param N the size of the input and  intermediate arrays
*/
bool gpu_parallel_reduction(scalar *input, scalar *intermediate, scalar *output, int helperIdx, int N)
    {
    unsigned int block_size = 256;
    unsigned int nblocks  = N/block_size + 1;
    //first do a block reduction of input
    unsigned int smem = block_size*sizeof(scalar);

    //Do a block reduction of the input array
    gpu_parallel_block_reduction2_kernel<<<nblocks,block_size,smem>>>(input,intermediate, N);
    HANDLE_ERROR(hipGetLastError());

    //sum reduce the temporary array, saving the result in the right slot of the output array
    gpu_serial_reduction_kernel<<<1,1>>>(intermediate,output,helperIdx,nblocks);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/*!
This serial reduction routine should probably never be called. It provides an interface to the
gpu_serial_reduction_kernel above that may be useful for testing
  */
bool gpu_serial_reduction(scalar *array, scalar *output, int helperIdx, int N)
    {
    gpu_serial_reduction_kernel<<<1,1>>>(array,output,helperIdx,N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/** @} */ //end of group declaration
