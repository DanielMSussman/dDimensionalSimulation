#include "hip/hip_runtime.h"
#include "utilities.cuh"
#include "functions.h"

/*! \file utilities.cu
  defines kernel callers and kernels for some simple GPU array calculations

 \addtogroup utilityKernels
 @{
 */

/*!
add the first N elements of array and put it in output[helperIdx]
*/
__global__ void gpu_serial_reduction_kernel(scalar *array, scalar *output, int helperIdx,int N)
    {
    scalar ans = 0.0;
    for (int i = 0; i < N; ++i)
        ans += array[i];
    output[helperIdx] = ans;
    return;
    };

/*!
perform a block reduction, storing the partial sums of input into output
*/
__global__ void gpu_parallel_block_reduction_kernel(scalar *input, scalar *output,int N)
    {
    extern __shared__ scalar sharedArray[];

    unsigned int tidx = threadIdx.x;
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    //load into shared memory and synchronize
    if(i < N)
        sharedArray[tidx] = input[i];
    else
        sharedArray[tidx] = 0.0;
    __syncthreads();

    //reduce
    for (int s = blockDim.x/2; s>0; s>>=1)
        {
        if (tidx < s)
            sharedArray[tidx] += sharedArray[tidx+s];
        __syncthreads();
        };
    //write to the correct block of the output array
    if (tidx==0)
        output[blockIdx.x] = sharedArray[0];
    };

/*!
a slight optimization of the previous block reduction, c.f. M. Harris presentation
*/
__global__ void gpu_parallel_block_reduction2_kernel(scalar *input, scalar *output,int N)
    {
    extern __shared__ scalar sharedArray[];

    unsigned int tidx = threadIdx.x;
    unsigned int i = 2*blockDim.x * blockIdx.x + threadIdx.x;

    scalar sum;
    //load into shared memory and synchronize
    if(i < N)
        sum = input[i];
    else
        sum = 0.0;
    if(i + blockDim.x < N)
        sum += input[i+blockDim.x];

    sharedArray[tidx] = sum;
    __syncthreads();

    //reduce
    for (int s = blockDim.x/2; s>0; s>>=1)
        {
        if (tidx < s)
            sharedArray[tidx] = sum = sum+sharedArray[tidx+s];
        __syncthreads();
        };
    //write to the correct block of the output array
    if (tidx==0)
        output[blockIdx.x] = sum;
    };

/*!
  A function of convenience...zero out an array on the device
  */
__global__ void gpu_zero_array_kernel(dVec *arr,
                                              int N)
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    dVec temp = make_dVec(0.0);
    arr[idx] = temp;
    for (int dd = 0; dd < DIMENSION; ++dd)
        arr[idx].x[dd] = 0.0;
    return;
    };
/*!
  A function of convenience...zero out an array on the device
  */
__global__ void gpu_zero_array_kernel(scalar *arr,
                                              int N)
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;

    arr[idx] = 0.;
    return;
    };
/*!
  A function of convenience...zero out an array on the device
  */
__global__ void gpu_zero_array_kernel(unsigned int *arr,
                                              int N)
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;

    arr[idx] = 0;
    return;
    };

/*!
  A function of convenience...zero out an array on the device
  */
__global__ void gpu_zero_array_kernel(int *arr,
                                      int N)
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;

    arr[idx] = 0;
    return;
    };

/*!
take a vector of dVecs, a vector of scalars, a factor, and return a vector where
every entry is 
factor*scalar[i]*(dVec[i])^2
*/
__global__ void gpu_scalar_times_dVec_squared_kernel(dVec *d_vec1, scalar *d_scalars, scalar factor, scalar *d_ans, int n)
    {
    // read in the index that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= n)
        return;
    d_ans[idx] = factor * d_scalars[idx]*dot(d_vec1[idx],d_vec1[idx]);
    };
/*!
take two vectors of dVecs and return a vector of scalars, where each entry is vec1[i].vec2[i]
*/
__global__ void gpu_dot_dVec_vectors_kernel(dVec *d_vec1, dVec *d_vec2, scalar *d_ans, int n)
    {
    // read in the index that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= n)
        return;
    d_ans[idx] = dot(d_vec1[idx],d_vec2[idx]);
    };
/*!
  multiply every element of an array of dVecs by the same scalar
  */
__global__ void gpu_dVec_times_scalar_kernel(dVec *d_vec1,scalar factor, int n)
    {
    // read in the index that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= n)
        return;
    d_vec1[idx] = factor*d_vec1[idx];
    };
/*!
  multiply every element of an array of dVecs by the same scalar
  */
__global__ void gpu_dVec_times_scalar_kernel(dVec *d_vec1,scalar factor, dVec *d_ans,int n)
    {
    // read in the index that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= n)
        return;
    d_ans[idx] = factor*d_vec1[idx];
    };

/////
//Kernel callers
///

/*!
\param d_vec1 dVec input array
\param factor scalar multiplication factor
\param N      the length of the arrays
\post d_vec1 *= factor for every element
 */
bool gpu_dVec_times_scalar(dVec *d_vec1, scalar factor, int N)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;
    gpu_dVec_times_scalar_kernel<<<nblocks,block_size>>>(
                                                d_vec1,
                                                factor,
                                                N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };
bool gpu_dVec_times_scalar(dVec *d_vec1, scalar factor, dVec *d_ans,int N)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;
    gpu_dVec_times_scalar_kernel<<<nblocks,block_size>>>(
                                                d_vec1,
                                                factor,
                                                d_ans,
                                                N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

bool gpu_scalar_times_dVec_squared(dVec *d_vec1, scalar *d_scalars, scalar factor, scalar *d_ans, int N)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;
    gpu_scalar_times_dVec_squared_kernel<<<nblocks,block_size>>>(
                                                d_vec1,
                                                d_scalars,
                                                factor,
                                                d_ans,
                                                N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/*!
\param d_vec1 dVec input array
\param d_vec2 dVec input array
\param d_ans  scalar output array... d_ans[idx] = d_vec1[idx].d_vec2[idx]
\param N      the length of the arrays
\post d_ans = d_vec1.d_vec2
*/
bool gpu_dot_dVec_vectors(dVec *d_vec1, dVec *d_vec2, scalar *d_ans, int N)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;
    gpu_dot_dVec_vectors_kernel<<<nblocks,block_size>>>(
                                                d_vec1,
                                                d_vec2,
                                                d_ans,
                                                N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

bool gpu_zero_array(dVec *arr,
                    int N
                    )
    {
    //optimize block size later
    unsigned int block_size = 128;
    if (N < 128) block_size = 16;
    unsigned int nblocks  = N/block_size + 1;

    gpu_zero_array_kernel<<<nblocks, block_size>>>(arr,
                                                    N
                                                    );
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }

bool gpu_zero_array(unsigned int *arr,
                    int N
                    )
    {
    //optimize block size later
    unsigned int block_size = 128;
    if (N < 128) block_size = 16;
    unsigned int nblocks  = N/block_size + 1;

    gpu_zero_array_kernel<<<nblocks, block_size>>>(arr,
                                                    N
                                                    );
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }

bool gpu_zero_array(scalar *arr,
                    int N
                    )
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 16;
    unsigned int nblocks  = N/block_size + 1;

    gpu_zero_array_kernel<<<nblocks, block_size>>>(arr,
                                                    N
                                                    );
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }
bool gpu_zero_array(int *arr,
                    int N
                    )
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 16;
    unsigned int nblocks  = N/block_size + 1;

    gpu_zero_array_kernel<<<nblocks, block_size>>>(arr,
                                                    N
                                                    );
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }

/*!
a two-step parallel reduction algorithm that first does a partial sum reduction of input into the
intermediate array, then launches a second kernel to sum reduce intermediate into output[helperIdx]
\param input the input array to sum
\param intermediate an array that input is block-reduced to
\param output the intermediate array will be sum reduced and stored in one of the components of output
\param helperIdx the location in output to store the answer
\param N the size of the input and  intermediate arrays
*/
bool gpu_parallel_reduction(scalar *input, scalar *intermediate, scalar *output, int helperIdx, int N)
    {
    unsigned int block_size = 256;
    unsigned int nblocks  = N/block_size + 1;
    //first do a block reduction of input
    unsigned int smem = block_size*sizeof(scalar);

    //Do a block reduction of the input array
    gpu_parallel_block_reduction2_kernel<<<nblocks,block_size,smem>>>(input,intermediate, N);
    HANDLE_ERROR(hipGetLastError());

    //sum reduce the temporary array, saving the result in the right slot of the output array
    gpu_serial_reduction_kernel<<<1,1>>>(intermediate,output,helperIdx,nblocks);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

bool gpu_parallel_reduction(scalar *input, scalar *intermediate, scalar *output, int helperIdx, int N,int block_size)
    {
    unsigned int nblocks  = N/block_size + 1;
    //first do a block reduction of input
    unsigned int smem = block_size*sizeof(scalar);

    //Do a block reduction of the input array
    gpu_parallel_block_reduction2_kernel<<<nblocks,block_size,smem>>>(input,intermediate, N);
    HANDLE_ERROR(hipGetLastError());

    //sum reduce the temporary array, saving the result in the right slot of the output array
    gpu_serial_reduction_kernel<<<1,1>>>(intermediate,output,helperIdx,nblocks);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/*!
This serial reduction routine should probably never be called. It provides an interface to the
gpu_serial_reduction_kernel above that may be useful for testing
  */
bool gpu_serial_reduction(scalar *array, scalar *output, int helperIdx, int N)
    {
    gpu_serial_reduction_kernel<<<1,1>>>(array,output,helperIdx,N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/** @} */ //end of group declaration
