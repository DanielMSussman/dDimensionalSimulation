#include "hip/hip_runtime.h"
#include "velocityVerlet.cuh"
/*! \file velocityVerlet.cu 

\addtogroup updaterKernels
@{
*/

/*!
update the velocity in a velocity Verlet step
*/
__global__ void gpu_update_velocity_kernel(dVec *d_velocity, dVec *d_force, scalar deltaT, int n)
    {
    // read in the index that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= n)
        return;
    d_velocity[idx] += 0.5*deltaT*d_force[idx];
    };

/*!
calculate the displacement in a velocity verlet step according to the force and velocity
*/
__global__ void gpu_displacement_vv_kernel(dVec *d_displacement, dVec *d_velocity,
                                           dVec *d_force, scalar deltaT, int n)
    {
    // read in the index that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= n)
        return;
    d_displacement[idx] = deltaT*d_velocity[idx]+0.5*deltaT*deltaT*d_force[idx];
    };

/*!
\param d_velocity dVec array of velocity
\param d_force dVec array of force
\param deltaT time step
\param N      the length of the arrays
\post v = v + 0.5*deltaT*force
*/
bool gpu_update_velocity(dVec *d_velocity, dVec *d_force, scalar deltaT, int N)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;
    gpu_update_velocity_kernel<<<nblocks,block_size>>>(
                                                d_velocity,
                                                d_force,
                                                deltaT,
                                                N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/*!
\param d_displacement dVec array of displacements
\param d_velocity dVec array of velocities
\param d_force dVec array of forces
\param Dscalar deltaT the current time step
\param N      the length of the arrays
\post displacement = dt*velocity + 0.5 *dt^2*force
*/
bool gpu_displacement_velocity_verlet(dVec *d_displacement,
                      dVec *d_velocity,
                      dVec *d_force,
                      scalar deltaT,
                      int N)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;
    gpu_displacement_vv_kernel<<<nblocks,block_size>>>(
                                                d_displacement,d_velocity,d_force,deltaT,N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/** @} */ //end of group declaration
