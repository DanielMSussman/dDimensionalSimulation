#include "hip/hip_runtime.h"
#include "brownianDynamics.cuh"

/** \file brownianDynamics.cu
    * Defines kernel callers and kernels for GPU calculations of overdamped brownian dynamics
*/

/*!
    \addtogroup updaterKernels
    @{
*/

/*!
Each thread calculates the displacement of an individual cell
*/
__global__ void brownian_eom_integration_kernel(dVec *forces,
                                           dVec *displacements,
                                           hiprandState *RNGs,
                                           int N,
                                           scalar noisePrefactor,
                                           scalar forcePrefactor)
    {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >=N)
        return;
    hiprandState_t randState;

    randState=RNGs[idx];
    for (int dd = 0; dd < DIMENSION; ++dd)
        displacements[idx][dd] = noisePrefactor*cur_norm(&randState) + forcePrefactor*forces[idx][dd];

    RNGs[idx] = randState;
    return;
    };

//!get the current timesteps vector of displacements into the displacement vector
bool gpu_brownian_eom_integration(
                    dVec *forces,
                    dVec *displacements,
                    hiprandState *RNGs,
                    int N,
                    scalar deltaT,
                    scalar mu,
                    scalar T)
    {
    unsigned int block_size = 512;
    if (N < 512) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;

    scalar forcePrefactor = deltaT*mu;
    scalar noisePrefactor = sqrt(2.0*forcePrefactor*T);

    brownian_eom_integration_kernel<<<nblocks,block_size>>>(
                                forces,displacements,
                                RNGs,
                                N,noisePrefactor,forcePrefactor);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/** @} */ //end of group declaration

